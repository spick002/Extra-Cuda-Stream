/******************************************************************************
 *Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
//use pinned memory and buffers hipHostAlloc()
#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
    hipError_t cuda_ret;
    hipStream_t stream0,stream1,stream2;
   hipStreamCreate(&stream0);
   hipStreamCreate(&stream1);
   hipStreamCreate(&stream2);
   const unsigned int BLOCK_SIZE = 256;   
 // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d0, *B_d0, *C_d0;
    float *A_d1, *B_d1, *C_d1;
    float *A_d2, *B_d2, *C_d2;
    size_t A_sz, B_sz, C_sz, total_sz;
    unsigned VecSize;
   
    dim3 dim_grid, dim_block;

      if (argc == 1) {
        VecSize = 1000000;
      } 
      else if (argc == 2) {
      VecSize = atoi(argv[1]);     
      }
      else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
      }

    A_sz = 333333;
    B_sz = 333333;
    C_sz = 333333;
    total_sz=1000000;
    hipHostAlloc( (void **) &A_h, sizeof(float)*total_sz,hipHostMallocDefault );
    for (unsigned int i=0; i < total_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    hipHostAlloc( (void **) &B_h, sizeof(float)*total_sz,hipHostMallocDefault );
    for (unsigned int i=0; i < total_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    hipHostAlloc( (void **) &C_h, sizeof(float)*total_sz,hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize);

    // Allocate device variables ----------------------------------------------
//tell the GPU/device how much of its (the device's) memory to allocate
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
	hipMalloc((void**) &A_d0,4*A_sz);
	hipMalloc((void**) &B_d0,4*B_sz);
	hipMalloc((void**) &C_d0,4*C_sz);
        hipMalloc((void**) &A_d1,4*A_sz);
        hipMalloc((void**) &B_d1,4*B_sz);
        hipMalloc((void**) &C_d1,4*C_sz);
        hipMalloc((void**) &A_d2,4*(A_sz+1));//need to cover all 1000000 in vectors
        hipMalloc((void**) &B_d2,4*(B_sz+1));
        hipMalloc((void**) &C_d2,4*(C_sz+1));

    //hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    //startTime(&timer);

    //INSERT CODE HERE
	hipMemcpyAsync(A_d0,A_h,4*A_sz,hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(B_d0,B_h,4*B_sz,hipMemcpyHostToDevice,stream0);
	VecAdd<<<(A_sz-1)/BLOCK_SIZE+1,BLOCK_SIZE,0,stream0>>>(A_sz,A_d0,B_d0,C_d0);
	//hipDeviceSynchronize();
	
	hipMemcpyAsync(A_d1,A_h+A_sz,4*A_sz,hipMemcpyHostToDevice,stream1);
        hipMemcpyAsync(B_d1,B_h+B_sz,4*B_sz,hipMemcpyHostToDevice,stream1);
	VecAdd<<<(A_sz-1)/BLOCK_SIZE+1,BLOCK_SIZE,0,stream1>>>(A_sz,A_d1,B_d1,C_d1);
	//hipDeviceSynchronize();	

	hipMemcpyAsync(A_d2,A_h+2*A_sz,4*(A_sz+1),hipMemcpyHostToDevice,stream2);
        hipMemcpyAsync(B_d2,B_h+2*B_sz,4*(B_sz+1),hipMemcpyHostToDevice,stream2);
	VecAdd<<<(A_sz+1-1)/BLOCK_SIZE+1,BLOCK_SIZE,0,stream2>>>(A_sz+1,A_d2,B_d2,C_d2);
	//hipDeviceSynchronize();

//	hipStreamSynchronize(stream0);
//	hipStreamSynchronize(stream1);
//	hipStreamSynchronize(stream2);
    
//hipDeviceSynchronize();
    //stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel  ---------------------------
    //printf("Launching kernel..."); fflush(stdout);
    //startTime(&timer);
    //cuda_ret = hipDeviceSynchronize();
//	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    //stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Copy device variables from host ----------------------------------------

    //printf("Copying data from device to host..."); fflush(stdout);
    //startTime(&timer);

    //INSERT CODE HERE
	hipMemcpyAsync(C_h,C_d0,4*C_sz,hipMemcpyDeviceToHost,stream0);
 	hipMemcpyAsync(C_h+C_sz,C_d1,4*C_sz,hipMemcpyDeviceToHost,stream1);
	hipMemcpyAsync(C_h+2*C_sz,C_d2,4*(C_sz+1),hipMemcpyDeviceToHost,stream2);

        hipStreamSynchronize(stream0);
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
    //hipDeviceSynchronize();
    //stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    //printf("Verifying results..."); fflush(stdout);

    //verify(A_h, B_h, C_h, VecSize);


    // Free memory ------------------------------------------------------------

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);

    //INSERT CODE HERE
	hipFree(A_d0);
	hipFree(B_d0);
	hipFree(C_d0);
	hipFree(A_d1);
        hipFree(B_d1);
        hipFree(C_d1);
        hipFree(A_d2);
        hipFree(B_d2);
        hipFree(C_d2);

    return 0;

}
